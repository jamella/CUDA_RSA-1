#include "hip/hip_runtime.h"
\*
 * C++ CUDE file containing RSA Parallel Code
 *
 * Copyright 2015 Vedsar
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*\

#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#define TICK gettimeofday(&start,NULL); 
#define TOCK gettimeofday(&end,NULL); 
#define TIME ((end.tv_sec-start.tv_sec)*1000000+(end.tv_usec-start.tv_usec))
#define sz 100
__device__ int temp_mat[sz][2*sz+1];
__device__ int cpy_p[sz];
__device__ int half_carr[sz];
__device__ int rem_mul[sz];
__device__ __shared__ int flag;
__device__ __shared__ int tf1;

__global__ void cuda_prime(int *a,int *p,int lena,int lenp,int sz) {
	int r_len,len;
	int i,num,c_temp,tf;
	while(1) {
		if(threadIdx.x==0) {
			flag=0;
			if(p[lenp-1]%2==0) {
				flag=1;
			}
		}
		__syncthreads();
		if(flag==1) {
			//do square a^2 mod p
			num=p[blockIdx.x];
			c_temp=num*p[threadIdx.x];
			temp_mat[blockIdx.x][threadIdx.x]=c_temp;
			//check carry
			tf=1;
			while(tf==1) {
				tf=0;
				if(temp_mat[blockIdx.x][threadIdx.x]>9) {
					tf=1;
					temp_mat[blockIdx.x][threadIdx.x+1]+=(c_temp/10);
					temp_mat[blockIdx.x][threadIdx.x]=c_temp%10;
				}
			}
			//half cpy_p
			if(blockIdx.x==0) {
				if(cpy_p[threadIdx.x]%2!=0)
					half_carr[threadIdx.x]=1;
				cpy_p[threadIdx.x]=cpy_p[threadIdx.x]/2;
				if(half_carr[threadIdx.x]==1)
					cpy_p[threadIdx.x-11]+=5;
			
		}
		else {
			//do multi
			if(len==0) {
				if(blockIdx.x==0) {
					rem_mul[threadIdx.x]=cpy_p[threadIdx.x];
					if(threadIdx.x==0) {
						if(cpy_p[0]==0) {
							cpy_p[0]=9;
							cpy_p[1]=cpy_p[1]-1;
						}
						else {
							cpy_p[0]=cpy_p[0]-1;
						}
					}
				}
			}
			else {	
				//multi rem_mul with a mod p
				num=p[blockIdx.x];
	                        c_temp=num*p[threadIdx.x];
        	               	temp_mat[blockIdx.x][threadIdx.x]=c_temp;
                	        //check carry
                        	tf=1;
	                        while(tf==1) {
        	                        tf=0;
                	                if(temp_mat[blockIdx.x][threadIdx.x]>9) {
                        	               	tf=1;
                                	        temp_mat[blockIdx.x][threadIdx.x+1]+=(c_temp/10);
                                        	temp_mat[blockIdx.x][threadIdx.x]=c_temp%10;
	                               	}
        	                }

			}
		}
		//update length
		if(a[threadIdx.x]==-1)
			lena=threadIdx.x;
		if(cpy_p[threadIdx.x]==-1)
			lenp=threadIdx.x;
		//if(
		__syncthreads();
		if(lenp==1) {
			if(a[0]==1)
				fin[0]=1;
			else
				fin[0]=-1;
			break;
		}
	}
}

struct timeval start,end;

int main(int argc,char *argv[]) {
	int *dp,*mp,*mq,*dq,i,th,*da,*ma,*dfin,*mfin;
	mp=(int *)malloc(sz*sizeof(int));
	lenp=sz;
	lena=1;
	ma=(int *)malloc(sizeof(int)*sz);
	mfin=(int *)malloc(sizeof(int));
	hipMalloc(&dfin,sizeof(int));
	if(hipMalloc(&da,sizeof(int)*sz)!=hipSuccess) {
                printf("Not enough memory\n");
                return 0;
        }
	if(hipMalloc(&dp,sizeof(int)*sz)!=hipSuccess) {
		printf("Not enough memory\n");
		return 0;
	}
	//kernel call
	while(1) {
	for(i=0;i<sz;i++)
		mp[i]=rand()%10;

	hipMemcpy(mp,dp,sizeof(int)*sz,hipMemcpyHostToDevice);
	
	th=sz;
	th=((th/32)+1)*32;
	cuda_prime<<<sz,th>>>(da,dp,lena,lenp,sz);
	hipMemcpy(dfin,mfin,sizeof(int)*sz,hipMemcpyDeviceToHost);
	if(mfin[0]==1)
		break;
	}
	}
	mq=(int *)malloc(sz*sizeof(int));
        hipMalloc(&dq,sizeof(int)*sz);
	while(1) {
	for(i=0;i<sz;i++)
                mq[i]=rand()%10;
        
        hipMemcpy(mq,dq,sizeof(int)*sz,hipMemcpyHostToDevice);
       	
	th=sz;
        th=((th/32)+1)*32;
        cuda_test<<<sz,th>>>(dq,sz);
	
	//hipMemcpy(dq,mq,sizeof(int)*sz,hipMemcpyHostToDevice);
	hipMemcpy(dfin,mfin,sizeof(int)*sz,hipMemcpyDeviceToHost);
	if(mfin[0]==1)
		break;
	}
	}
	return 0;
}
